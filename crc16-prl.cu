#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

static const uint16_t table[256] = {
    0x0000, 0x1021, 0x2042, 0x3063, 0x4084, 0x50A5, 0x60C6, 0x70E7, 0x8108, 0x9129, 0xA14A, 0xB16B, 0xC18C, 0xD1AD, 0xE1CE, 0xF1EF,
0x1231, 0x0210, 0x3273, 0x2252, 0x52B5, 0x4294, 0x72F7, 0x62D6, 0x9339, 0x8318, 0xB37B, 0xA35A, 0xD3BD, 0xC39C, 0xF3FF, 0xE3DE,
0x2462, 0x3443, 0x0420, 0x1401, 0x64E6, 0x74C7, 0x44A4, 0x5485, 0xA56A, 0xB54B, 0x8528, 0x9509, 0xE5EE, 0xF5CF, 0xC5AC, 0xD58D,
0x3653, 0x2672, 0x1611, 0x0630, 0x76D7, 0x66F6, 0x5695, 0x46B4, 0xB75B, 0xA77A, 0x9719, 0x8738, 0xF7DF, 0xE7FE, 0xD79D, 0xC7BC,
0x48C4, 0x58E5, 0x6886, 0x78A7, 0x0840, 0x1861, 0x2802, 0x3823, 0xC9CC, 0xD9ED, 0xE98E, 0xF9AF, 0x8948, 0x9969, 0xA90A, 0xB92B,
0x5AF5, 0x4AD4, 0x7AB7, 0x6A96, 0x1A71, 0x0A50, 0x3A33, 0x2A12, 0xDBFD, 0xCBDC, 0xFBBF, 0xEB9E, 0x9B79, 0x8B58, 0xBB3B, 0xAB1A,
0x6CA6, 0x7C87, 0x4CE4, 0x5CC5, 0x2C22, 0x3C03, 0x0C60, 0x1C41, 0xEDAE, 0xFD8F, 0xCDEC, 0xDDCD, 0xAD2A, 0xBD0B, 0x8D68, 0x9D49,
0x7E97, 0x6EB6, 0x5ED5, 0x4EF4, 0x3E13, 0x2E32, 0x1E51, 0x0E70, 0xFF9F, 0xEFBE, 0xDFDD, 0xCFFC, 0xBF1B, 0xAF3A, 0x9F59, 0x8F78,
0x9188, 0x81A9, 0xB1CA, 0xA1EB, 0xD10C, 0xC12D, 0xF14E, 0xE16F, 0x1080, 0x00A1, 0x30C2, 0x20E3, 0x5004, 0x4025, 0x7046, 0x6067,
0x83B9, 0x9398, 0xA3FB, 0xB3DA, 0xC33D, 0xD31C, 0xE37F, 0xF35E, 0x02B1, 0x1290, 0x22F3, 0x32D2, 0x4235, 0x5214, 0x6277, 0x7256,
0xB5EA, 0xA5CB, 0x95A8, 0x8589, 0xF56E, 0xE54F, 0xD52C, 0xC50D, 0x34E2, 0x24C3, 0x14A0, 0x0481, 0x7466, 0x6447, 0x5424, 0x4405,
0xA7DB, 0xB7FA, 0x8799, 0x97B8, 0xE75F, 0xF77E, 0xC71D, 0xD73C, 0x26D3, 0x36F2, 0x0691, 0x16B0, 0x6657, 0x7676, 0x4615, 0x5634,
0xD94C, 0xC96D, 0xF90E, 0xE92F, 0x99C8, 0x89E9, 0xB98A, 0xA9AB, 0x5844, 0x4865, 0x7806, 0x6827, 0x18C0, 0x08E1, 0x3882, 0x28A3,
0xCB7D, 0xDB5C, 0xEB3F, 0xFB1E, 0x8BF9, 0x9BD8, 0xABBB, 0xBB9A, 0x4A75, 0x5A54, 0x6A37, 0x7A16, 0x0AF1, 0x1AD0, 0x2AB3, 0x3A92,
0xFD2E, 0xED0F, 0xDD6C, 0xCD4D, 0xBDAA, 0xAD8B, 0x9DE8, 0x8DC9, 0x7C26, 0x6C07, 0x5C64, 0x4C45, 0x3CA2, 0x2C83, 0x1CE0, 0x0CC1,
0xEF1F, 0xFF3E, 0xCF5D, 0xDF7C, 0xAF9B, 0xBFBA, 0x8FD9, 0x9FF8, 0x6E17, 0x7E36, 0x4E55, 0x5E74, 0x2E93, 0x3EB2, 0x0ED1, 0x1EF0,
};

const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__global__
void crc16kernel(uint8_t* data, uint16_t crc, int length) {
    /// YOUR CODE
    //int x = blockIdx.x * blockDim.x + threadIdx.x;
    //int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    

    while (length--) {
        crc = table[((crc>>8) ^ *data++) ] ^ (crc<<8);
    }
    //lasciare in device l'array data e fare uno shift ogni volta che entro nella kernel
}

__global__
void xorkernel(uint8_t* data, uint16_t crc, int length) {
    /// YOUR CODE
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    

}

const int N = 300;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    uint16_t hcrc = 0xffff;
    uint8_t hdata[] = "ciaociao";

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    
    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for(int i = 0; i < sizeof(hdata); i++) {
        hcrc = table[((hcrc>>8) ^ hdata[i])] ^ (hcrc<<8);
    }

    TM_host.stop();
    TM_host.print("MatrixMultiplication host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    TM_device.start();

    uint16_t dcrc = 0xffff;
    uint8_t ddata[] = "ciaociao";
    int length = sizeof(ddata);
    SAFE_CALL(hipMalloc(&dcrc, sizeof(uint16_t)));
    SAFE_CALL(hipMalloc(&ddata, length*sizeof(uint8_t));

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVICE
    SAFE_CALL(hipMemcpy(dcrc, 0xffff, sizeof(uint16_t), hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(ddata, "ciaociao", length*sizeof(uint8_t), hipMemcpyHostToDevice));

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    //TM_device.start();

    dim3 block_size(32, 32, 1);
    dim3 num_blocks(ceil((float)N/32), ceil((float)N/32), 1);
    int n_temp = length;
    int n_iter = 0;

    while(n_temp > 1) {
        n_temp /= 16;
        n_iter++;
    }

    int tile_temp = length;
    int block = 256*2;

    for(int i = 0; i < n_iter; i++){
        crc16kernel<<<ceil((float)lenght/tile_temp), 256>>>(ddata, dcrc, length);
        tile_temp *= block;
    }
    //crc16kernel<<< num_blocks, block_size >>>(ddata, dcrc, length);

    /*TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("MatrixMultiplication device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";*/

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    hipDeviceSynchronize();
    SAFE_CALL(hipMemcpy(h_crc_tmp, dcrc, sizeof(uint16_t), hipMemcpyDeviceToHost));

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("MatrixMultiplication device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // RESULT CHECK
    
    if (hcrc != h_crc_tmp) {
        /*std::cerr << "wrong result at: ("
                << (i / N) << ", " << (i % N) << ")"
                << "\nhost:   " << h_matrixC[i]
                << "\ndevice: " << h_matrix_tmp[i] << "\n\n";*/
        std::cerr << "wrong result\n\n"
        hipDeviceReset();
        std::exit(EXIT_FAILURE);
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] hdata;
    delete[] hcrc;
    delete[] h_crc_temp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL(hipFree(ddata));
    SAFE_CALL(hipFree(dcrc));

    // -------------------------------------------------------------------------
    //hipDeviceReset();
}
