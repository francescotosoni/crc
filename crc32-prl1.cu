#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <iomanip>
#include <fstream>
#include <vector>
#include <string>
#include <random>
#include "tables.hpp"
#include "Timer.cuh"
#include "CheckError.cuh"

using namespace timer;

__global__
void crc32kernel(uint8_t* data, uint32_t* crc, int length, uint32_t* d_table, uint32_t* tmp) {
    /// YOUR CODE
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    *crc = 0xffffffff;
    uint32_t* current = (uint32_t*)data;
    
    if(id < 4) { 
        uint32_t one = *current ^ *crc;
        int i = (256*((length-1)-id))+((one>>(8*id)) & 0xff);
        tmp[id] = d_table[i];
    }
    else if(id > 3 && id < length) {
        uint32_t two = *(current+(id / 4));
        int i = (256*((length-1)-id))+(two>>(8*(id % 4)) & 0xff);
        tmp[id] = d_table[i];
    }

    __syncthreads();

    if(id == length+5) {
        crc[0] = tmp[0];
        for(int i = 1; i < length; i++) {
            crc[0] ^= (tmp[i]);
        }
        crc[0] ^= 0xffffffff;
    }
}

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;

    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
   
    uint32_t hcrc = 0xffffffff;
    std::ifstream fin("input.txt");
    std::string temp;
    std::string d("");

    while(fin.good()){
        fin >> temp;
        d.append(temp);
    }
    
    std::cout << d.length() << std::endl;
    uint8_t data[d.length()];

    for(int i=0;i<d.length();i++){
        data[i] = d[i] - '0';
    }

    int length = sizeof(data);
    //printf("%d\n", length);

    //uint32_t table[length][256];
    auto *table = (uint32_t *)malloc(length * 256 * sizeof(uint32_t));
    
    make_crc_table_reverse(table, length);
    
    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    
    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();
    //std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();

    for(int i = 0; i < length; i++) {
        hcrc = table[(hcrc ^ data[i]) & 0xFF] ^ (hcrc>>8);
    }

    hcrc ^= 0xffffffff;

    TM_host.stop();
    TM_host.print("CRC32C host:   ");
    //std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    //double tempo = std::chrono::duration_cast<std::chrono::duration<double> >(end - start).count();
    //std::cout << "Panato cpu time:" << tempo << std::endl;
    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    TM_device.start();

    uint32_t crc = 0xffffffff;

    uint32_t* dcrc;
    uint8_t* ddata;
    uint32_t* d_table;
    uint32_t *tmp;
    hipMalloc(&dcrc, length*sizeof(uint32_t));
    hipMalloc(&ddata, length*sizeof(uint8_t));
    hipMalloc(&d_table, length * 256 * sizeof(uint32_t));
    hipMalloc(&tmp, length * sizeof(uint32_t));

    
    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVICE
    hipMemcpy(ddata, data, length*sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_table, table, length * 256 * sizeof(uint32_t), hipMemcpyHostToDevice);
    
    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    //TM_device.start();

    dim3 block_size(1024, 1, 1);
    dim3 num_blocks(ceil((float)length/1024), 1, 1);
   
    crc32kernel<<< num_blocks, block_size >>>(ddata, dcrc, length, d_table, tmp);
    hipDeviceSynchronize();

    /*TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("CRC32C device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";*/

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    uint32_t h_crc_tmp;
    hipMemcpy(&h_crc_tmp, dcrc, sizeof(uint32_t), hipMemcpyDeviceToHost);

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("CRC32 device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // RESULT CHECK
    printf("0x%x - 0x%x\n", hcrc, h_crc_tmp);
    
    if (hcrc != h_crc_tmp) {
        /*std::cerr << "wrong result at: ("
                << (i / N) << ", " << (i % N) << ")"
                << "\nhost:   " << h_matrixC[i]
                << "\ndevice: " << h_matrix_tmp[i] << "\n\n";*/
        //std::cerr << "wrong result\n\n";
        //hipDeviceReset();
        std::exit(EXIT_FAILURE);
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    //delete[] hdata;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    hipFree(d_table);
    hipFree(ddata);
    hipFree(dcrc);

    // -------------------------------------------------------------------------
    //hipDeviceReset();
}
